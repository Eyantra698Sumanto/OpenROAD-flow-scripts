#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////
//
// BSD 3-Clause License
//
// Copyright (c) 2019, The Regents of the University of California
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
///////////////////////////////////////////////////////////////////////////////

#include <cusp/precond/diagonal.h>
#include <cusp/blas/blas.h>
#include <cusp/krylov/bicgstab.h>

#include "gpuSolver.h"

namespace gpl {
using namespace std;
using utl::GPL;



void GpuSolver::cudaerror(hipError_t code)
{
  if (code != hipSuccess) {
    log_->error(GPL,
                1,
                "[CUDA ERROR] {} at line {} in file {} \n",
                hipGetErrorString(code),
                __LINE__,
                __FILE__);
  }
}
void GpuSolver::cusparseerror(hipsparseStatus_t code)
{
  if (code != HIPSPARSE_STATUS_SUCCESS) {
    log_->error(GPL,
                1,
                "[CUSPARSE ERROR] {} at line {} in file {}\n",
                hipsparseGetErrorString(code),
                __LINE__,
                __FILE__);
  }
}

void GpuSolver::cusolvererror(hipsolverStatus_t code)
{
  if (code != HIPSOLVER_STATUS_SUCCESS) {
    log_->error(GPL,
                1,
                "[CUSOLVER ERROR] {} at line {} in file {}\n",
                hipGetErrorString(*(hipError_t*) &code),
                __LINE__,
                __FILE__);
  }
}

GpuSolver::GpuSolver(SMatrix& placeInstForceMatrix,
                     Eigen::VectorXf& fixedInstForceVec,
                     utl::Logger* logger)
{
  // Convert the sparse matrix to COO format on the host
  const int nnz = placeInstForceMatrix.nonZeros();
  vector<int> cooRowIndex(nnz), cooColIndex(nnz);
  vector<float> cooVal(nnz);
  int cnt = 0;
  for (int row = 0; row < placeInstForceMatrix.outerSize(); row++) {
    for (typename SMatrix::InnerIterator it(placeInstForceMatrix, row); it; ++it) {
      cooRowIndex[cnt] = it.row();
      cooColIndex[cnt] = it.col();
      cooVal[cnt] = it.value();
      cnt++;
    }
  }

  // Copy the COO formatted triplets to device
  d_cooRowIndex_ = cooRowIndex;
  d_cooColIndex_ = cooColIndex;
  d_cooVal_ = cooVal;
  d_instLocVec_.resize(fixedInstForceVec.size());

  // Set raw pointers to point to the triplets in the device
  r_cooRowIndex_ = d_cooRowIndex_.data();
  r_cooColIndex_ = d_cooColIndex_.data();
  r_cooVal_ = d_cooVal_.data();
  r_instLocVec_ = d_instLocVec_.data();

  m_ = fixedInstForceVec.size();
  log_ = logger;
}


void GpuSolver::cusolverCal(Eigen::VectorXf& instLocVec)
{
  // Updated CUDA solver using CUSP library
  thrust::device_ptr<int> p_rowInd
      = thrust::device_pointer_cast(r_cooRowIndex_);
  thrust::device_ptr<int> p_colInd
      = thrust::device_pointer_cast(r_cooColIndex_);
  thrust::device_ptr<float> p_val = thrust::device_pointer_cast(r_cooVal_);
  thrust::device_ptr<float> d_fixedInstForceVec_
      = thrust::device_pointer_cast(r_fixedInstForceVec_);
  thrust::device_ptr<float> p_instLocVec_ = thrust::device_pointer_cast(r_instLocVec_);

  // use array1d_view to wrap the individual arrays
  typedef typename cusp::array1d_view<thrust::device_ptr<int>>
      DeviceIndexArrayView;
  typedef typename cusp::array1d_view<thrust::device_ptr<float>>
      DeviceValueArrayView;
  DeviceIndexArrayView row_indices(p_rowInd, p_rowInd + nnz_);
  DeviceIndexArrayView column_indices(p_colInd, p_colInd + nnz_);
  DeviceValueArrayView values(p_val, p_val + nnz_);
  DeviceValueArrayView d_x(p_instLocVec_, p_instLocVec_ + m_);
  DeviceValueArrayView d_b(d_fixedInstForceVec_, d_fixedInstForceVec_ + m_);

  // combine the three array1d_views into a coo_matrix_view
  typedef cusp::coo_matrix_view<DeviceIndexArrayView,
                                DeviceIndexArrayView,
                                DeviceValueArrayView>
      DeviceView;

  // construct a coo_matrix_view from the array1d_views
  DeviceView d_A(m_, m_, nnz_, row_indices, column_indices, values);

  // set stopping criteria.
  int iteration_limit = 100;
  float relative_tolerance = 1e-15;
  bool verbose = false;  // Decide if the CUDA solver prints the iteration
                         // details or not.
  cusp::monitor<float> monitor_(
      d_b, iteration_limit, relative_tolerance, verbose);

  // setup preconditioner
  cusp::precond::diagonal<float, cusp::device_memory> d_M(d_A);

  // solve the linear system A * x = b with the BICGSTAB method
  cusp::krylov::bicgstab(d_A, d_x, d_b, monitor_, d_M);

  // Sync and Copy data to host
  cudaerror(hipMemcpy(instLocVec.data(),
                       r_instLocVec_,
                       sizeof(float) * m_,
                       hipMemcpyDeviceToHost));

  // Calculate  AX = A * X - B
  cusp::coo_matrix<int, float, cusp::device_memory> A(d_A);
  cusp::array1d<float, cusp::device_memory> X(d_x);
  cusp::array1d<float, cusp::device_memory> B(d_b);
  cusp::array1d<float, cusp::device_memory> AX(m_);
  cusp::multiply(A, X, AX);
  cusp::blas::axpy(B, AX, -1);

  // Calculate L1 norm of the residual vector.
  error_ = cusp::blas::nrm1(AX) / cusp::blas::nrm1(B);
}

float GpuSolver::error()
{
  return (error_ > 0) ? error_ : -error_;
}

GpuSolver::~GpuSolver()
{
}
}  // namespace gpl
